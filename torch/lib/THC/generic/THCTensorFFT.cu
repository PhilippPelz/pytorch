
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorFFT.cu"
#else

#if defined(THC_REAL_IS_ZFLOAT) || defined(THC_REAL_IS_ZDOUBLE)

// THC_API int THCState_getNumCuFFTPlans(THCState* state);
// THC_API cufftHandle* THCState_getCuFFTPlan(THCState* state,int batch, int n1, int n2, int n3) ;
void THCTensor_(fftnbase)(THCState *state, THCTensor *self, THCTensor *result, int direction) {
	int ndim = THCTensor_(nDimension)(state, result);
	int batch = 1;
	int *fft_dims = (int*)malloc(ndim*sizeof(int));
	for (int i = 0; i < ndim; i++) {
		fft_dims[i] = (int) THCTensor_(size)(state, self, i);
	}
	cufftHandle plan;
	cufftSafeCall(cufftPlanMany(&plan, ndim, fft_dims, NULL, 1, 0, NULL, 1, 0, cufftname, batch));
	cufftSafeCall(cufftSetStream(plan, THCState_getCurrentStream(state)));
	cufftSafeCall(cufft(plan, (cureal *)THCTensor_(data)(state, self), (cureal *)THCTensor_(data)(state, result), direction));
	cufftDestroy(plan);
	free(fft_dims);
}

// takes the first dimension as batch dimension
void THCTensor_(fftnBatchedbase)(THCState *state, THCTensor *self, THCTensor *result, int direction) {
	int ndim = THCTensor_(nDimension)(state, self) -1;
	int batch = THCTensor_(size)(state, self, 0);
	int *fft_dims = (int*)malloc(ndim*sizeof(int));
	// FILE *f;
	// printf("ndim = %d\n",ndim);
	// printf("batch = %d\n",batch);
	// printf("in fftnBatchedbase\n");
	//f = fopen("/home/philipp/fftnBatchedbase.log", "a+");
	//fprintf(f,"fftnBatchedbase start" );
	int dist =1;
	for (int i = 1; i <= ndim ; i++) {
		fft_dims[i - 1] = (int) THCTensor_(size)(state, self, i);
		dist *= fft_dims[i - 1];
	}
	cufftHandle handle;
	cufftSafeCall(cufftPlanMany(&handle, ndim, fft_dims, NULL, 1, dist, NULL, 1, dist, cufftname, batch));
	//fprintf(f,"cufftPlanMany\n");
	// printf("cufftPlanMany\n");
	cufftSafeCall(cufftSetStream(handle, THCState_getCurrentStream(state)));
	// printf("cufftSetStream\n");
	cufftSafeCall(cufft(handle, (cureal *)THCTensor_(data)(state, self), (cureal *)THCTensor_(data)(state, result), direction));
	// printf("cufft\n");
	cufftDestroy(handle);

	free(fft_dims);
}

void THCTensor_(fftnBatched)(THCState *state, THCTensor *self, THCTensor *result) {
	THCTensor_(fftnBatchedbase)(state, self, result, CUFFT_FORWARD);
	int m = THCTensor_(nElement)(state, result);
	m /= THCTensor_(size)(state, result,0);
	THCTensor_(mul)(state, result, result, ccx(1 / sqrt(m)));
}

void THCTensor_(ifftnBatched)(THCState *state, THCTensor *self, THCTensor *result) {
	THCTensor_(fftnBatchedbase)(state, self, result, CUFFT_INVERSE);
	int m = THCTensor_(nElement)(state, result);
	m /= THCTensor_(size)(state, result,0);
	THCTensor_(mul)(state, result, result, ccx(1 / sqrt(m),0));
}

void THCTensor_(fft)(THCState *state, THCTensor *result, THCTensor *self) {
	THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, result, self));
	if (self != result)
		THCTensor_(resizeAs)(state, result, self);
	if(THCTensor_(nDimension)(state, self) < 1)
		THError("tensor must at least have dimension 1\n");
	int self_ndim = THCTensor_(nDimension)(state, self);
	if (!THCTensor_(isSameSizeAs)(state, self, result))
    THError("self_ndim must be equal result_ndim\n");
	int self_batch_dim = 1;
	for(int i = 0; i< self_ndim-1; i++){
		self_batch_dim *= THCTensor_(size)(state, self, i);
	}

	THLongStorage *new_self_size = THLongStorage_newWithSize2( self_batch_dim, THCTensor_(size)(state, self, self_ndim-1));
	THCTensor *new_self = THCTensor_(newView)(state, self, new_self_size);

	THLongStorage *new_result_size = THLongStorage_newWithSize2( self_batch_dim, THCTensor_(size)(state, self, self_ndim-1));
	THCTensor *new_result = THCTensor_(newView)(state, result, new_result_size);

	THCTensor_(fftnBatched)(state,new_self,new_result);
	THLongStorage_free(new_self_size);
	THLongStorage_free(new_result_size);
	THCTensor_(free)(state,new_result);
	THCTensor_(free)(state,new_self);
}
void THCTensor_(fft2)(THCState *state, THCTensor *result, THCTensor *self) {
	// printf("in fft2\n");
	THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, result, self));
	if (self != result)
		THCTensor_(resizeAs)(state, result, self);
	if(THCTensor_(nDimension)(state, self) < 2)
		THError("tensor must at least have dimension 2\n");
	int self_ndim = THCTensor_(nDimension)(state, self);
	int res_ndim = THCTensor_(nDimension)(state, result);
	// fprintf(f,"(self_dim,res_dim) = (%d,%d)\n",self_ndim,res_ndim);
	if (!THCTensor_(isSameSizeAs)(state, self, result))
    THError("self_ndim must be equal result_ndim\n");
	int self_batch_dim = 1;
	for(int i = 0; i< self_ndim-2; i++){
		self_batch_dim *= THCTensor_(size)(state, self, i);
	}
	// printf("self_batch_dim = %d\n",self_batch_dim);
  // printf("dim1 = %d\n",THCTensor_(size)(state, self, self_ndim-2));
	// printf("dim2 = %d\n",THCTensor_(size)(state, self, self_ndim-1));
	THLongStorage *new_self_size = THLongStorage_newWithSize3( self_batch_dim, THCTensor_(size)(state, self, self_ndim-2),THCTensor_(size)(state, self, self_ndim-1));
	// printf("after THLongStorage_newWithSize3\n");
	THCTensor *new_self = THCTensor_(newView)(state, self, new_self_size);
	// printf("after newView\n");

	THLongStorage *new_result_size = THLongStorage_newWithSize3( self_batch_dim, THCTensor_(size)(state, self, self_ndim-2),THCTensor_(size)(state, self, self_ndim-1));
	// printf("after THLongStorage_newWithSize3\n");
	THCTensor *new_result = THCTensor_(newView)(state, result, new_result_size);
	// printf("after newView\n");
	THCTensor_(fftnBatched)(state,new_self,new_result);
	THLongStorage_free(new_self_size);
	THLongStorage_free(new_result_size);
	THCTensor_(free)(state,new_result);
	THCTensor_(free)(state,new_self);
}

void THCTensor_(fft3)(THCState *state, THCTensor *result, THCTensor *self) {
	THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, result, self));
	if (self != result)
		THCTensor_(resizeAs)(state, result, self);
	if(THCTensor_(nDimension)(state, self) < 3)
		THError("tensor must at least have dimension 3\n");
	int self_ndim = THCTensor_(nDimension)(state, self);
	if (!THCTensor_(isSameSizeAs)(state, self, result))
    THError("self_ndim must be equal result_ndim\n");
	int self_batch_dim = 1;
	for(int i = 0; i< self_ndim-3; i++){
		self_batch_dim *= THCTensor_(size)(state, self, i);
	}

	THLongStorage *new_self_size = THLongStorage_newWithSize4( self_batch_dim, THCTensor_(size)(state, self, self_ndim-3),THCTensor_(size)(state, self, self_ndim-2),THCTensor_(size)(state, self, self_ndim-1));
	THCTensor *new_self = THCTensor_(newView)(state, self, new_self_size);

	THLongStorage *new_result_size = THLongStorage_newWithSize4( self_batch_dim, THCTensor_(size)(state, self, self_ndim-3),THCTensor_(size)(state, self, self_ndim-2),THCTensor_(size)(state, self, self_ndim-1));
	THCTensor *new_result = THCTensor_(newView)(state, result, new_result_size);

	THCTensor_(fftnBatched)(state,new_self,new_result);
	THLongStorage_free(new_self_size);
	THLongStorage_free(new_result_size);
	THCTensor_(free)(state,new_result);
	THCTensor_(free)(state,new_self);
}

void THCTensor_(fftn)(THCState *state, THCTensor *result, THCTensor *self) {
	THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, result, self));
	if (self != result)
		THCTensor_(resizeAs)(state, result, self);
	THCTensor_(fftnbase)(state, self, result, CUFFT_FORWARD);
	THCTensor_(mul)(state, result, result, ccx(1 / sqrt(THCTensor_(nElement)(state, result)),0));
}

void THCTensor_(ifft)(THCState *state, THCTensor *result, THCTensor *self) {
	THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, result, self));
	if (self != result)
		THCTensor_(resizeAs)(state, result, self);
	if(THCTensor_(nDimension)(state, self) < 1)
		THError("tensor must at least have dimension 1\n");
	int self_ndim = THCTensor_(nDimension)(state, self);
	if (!THCTensor_(isSameSizeAs)(state, self, result))
    THError("self_ndim must be equal result_ndim\n");
	int self_batch_dim = 1;
	for(int i = 0; i< self_ndim-1; i++){
		self_batch_dim *= THCTensor_(size)(state, self, i);
	}

	THLongStorage *new_self_size = THLongStorage_newWithSize2( self_batch_dim, THCTensor_(size)(state, self, self_ndim-1));
	THCTensor *new_self = THCTensor_(newView)(state, self, new_self_size);

	THLongStorage *new_result_size = THLongStorage_newWithSize2( self_batch_dim, THCTensor_(size)(state, self, self_ndim-1));
	THCTensor *new_result = THCTensor_(newView)(state, result, new_result_size);

	THCTensor_(ifftnBatched)(state,new_self,new_result);
	THLongStorage_free(new_self_size);
	THLongStorage_free(new_result_size);
	THCTensor_(free)(state,new_result);
	THCTensor_(free)(state,new_self);
}

void THCTensor_(ifft2)(THCState *state, THCTensor *result, THCTensor *self) {
	THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, result, self));
	if (self != result)
		THCTensor_(resizeAs)(state, result, self);
	if(THCTensor_(nDimension)(state, self) < 2)
		THError("tensor must at least have dimension 2\n");
	int self_ndim = THCTensor_(nDimension)(state, self);
	if (!THCTensor_(isSameSizeAs)(state, self, result))
    THError("self_ndim must be equal result_ndim\n");
	int self_batch_dim = 1;
	for(int i = 0; i< self_ndim-2; i++){
		self_batch_dim *= THCTensor_(size)(state, self, i);
	}

	THLongStorage *new_self_size = THLongStorage_newWithSize3( self_batch_dim, THCTensor_(size)(state, self, self_ndim-2),THCTensor_(size)(state, self, self_ndim-1));
	THCTensor *new_self = THCTensor_(newView)(state, self, new_self_size);

	THLongStorage *new_result_size = THLongStorage_newWithSize3( self_batch_dim, THCTensor_(size)(state, self, self_ndim-2),THCTensor_(size)(state, self, self_ndim-1));
	THCTensor *new_result = THCTensor_(newView)(state, result, new_result_size);

	THCTensor_(ifftnBatched)(state,new_self,new_result);
	THLongStorage_free(new_self_size);
	THLongStorage_free(new_result_size);
	THCTensor_(free)(state,new_result);
	THCTensor_(free)(state,new_self);
}

void THCTensor_(ifft3)(THCState *state, THCTensor *result, THCTensor *self) {
	THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, result, self));
	if (self != result)
		THCTensor_(resizeAs)(state, result, self);
	if(THCTensor_(nDimension)(state, self) < 3)
		THError("tensor must at least have dimension 3\n");
	int self_ndim = THCTensor_(nDimension)(state, self);
	if (!THCTensor_(isSameSizeAs)(state, self, result))
    THError("self_ndim must be equal result_ndim\n");
	int self_batch_dim = 1;
	for(int i = 0; i< self_ndim-3; i++){
		self_batch_dim *= THCTensor_(size)(state, self, i);
	}

	THLongStorage *new_self_size = THLongStorage_newWithSize4( self_batch_dim, THCTensor_(size)(state, self, self_ndim-3),THCTensor_(size)(state, self, self_ndim-2),THCTensor_(size)(state, self, self_ndim-1));
	THCTensor *new_self = THCTensor_(newView)(state, self, new_self_size);

	THLongStorage *new_result_size = THLongStorage_newWithSize4( self_batch_dim, THCTensor_(size)(state, self, self_ndim-3),THCTensor_(size)(state, self, self_ndim-2),THCTensor_(size)(state, self, self_ndim-1));
	THCTensor *new_result = THCTensor_(newView)(state, result, new_result_size);

	THCTensor_(ifftnBatched)(state,new_self,new_result);
	THLongStorage_free(new_self_size);
	THLongStorage_free(new_result_size);
	THCTensor_(free)(state,new_result);
	THCTensor_(free)(state,new_self);
}

void THCTensor_(ifftn)(THCState *state, THCTensor *result, THCTensor *self) {
	THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, result, self));
	if (self != result)
		THCTensor_(resizeAs)(state, result, self);
	THCTensor_(fftnbase)(state, self, result, CUFFT_INVERSE);
	THCTensor_(mul)(state, result, result, ccx(1 / sqrt(THCTensor_(nElement)(state, result)),0));
}

#endif
#endif
