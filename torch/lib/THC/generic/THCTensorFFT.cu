
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorFFT.cu"
#else

#if defined(THC_REAL_IS_ZFLOAT) || defined(THC_REAL_IS_ZDOUBLE)

// THC_API int THCState_getNumCuFFTPlans(THCState* state);
// THC_API cufftHandle* THCState_getCuFFTPlan(THCState* state,int batch, int n1, int n2, int n3) ;
void THCTensor_(fftnbase)(THCState *state, THCTensor *self, THCTensor *result, int direction) {
	int ndim = THCTensor_(nDimension)(state, result);
	int batch = 1;
	int *fft_dims = (int*)malloc(ndim*sizeof(int));
	for (int i = 0; i < ndim; i++) {
		fft_dims[i] = (int) THCTensor_(size)(state, self, i);
	}
	cufftHandle plan;
	cufftSafeCall(cufftPlanMany(&plan, ndim, fft_dims, NULL, 1, 0, NULL, 1, 0, cufftname, batch));
	cufftSafeCall(cufftSetStream(plan, THCState_getCurrentStream(state)));
	cufftSafeCall(cufft(plan, (cureal *)THCTensor_(data)(state, self), (cureal *)THCTensor_(data)(state, result), direction));
	cufftDestroy(plan);
	free(fft_dims);
}

// takes the first dimension as batch dimension
void THCTensor_(fftnBatchedbase)(THCState *state, THCTensor *self, THCTensor *result, int direction) {
	int ndim = THCTensor_(nDimension)(state, self) -1;
	int batch = THCTensor_(size)(state, self, 0);
	int *fft_dims = (int*)malloc(ndim*sizeof(int));
	FILE *f;
	//f = fopen("/home/philipp/fftnBatchedbase.log", "a+");
	//fprintf(f,"fftnBatchedbase start" );
	for (int i = 1; i <= ndim ; i++) {
		fft_dims[i - 1] = (int) THCTensor_(size)(state, self, i);
		//fprintf(f,"fft_dims[i - 1] = %d",fft_dims[i - 1]);
	}
	cufftHandle handle;
	cufftSafeCall(cufftPlanMany(&handle, ndim, fft_dims, NULL, 1, 0, NULL, 1, 0, cufftname, batch));
	//fprintf(f,"cufftPlanMany\n");
	//fclose(f);
	cufftSafeCall(cufftSetStream(handle, THCState_getCurrentStream(state)));
	//fprintf(f,"cufftSetStream\n");
	cufftSafeCall(cufft(handle, (cureal *)THCTensor_(data)(state, self), (cureal *)THCTensor_(data)(state, result), direction));
	//fprintf(f,"cufft\n");
	cufftDestroy(handle);

	free(fft_dims);
}

void THCTensor_(fftnBatched)(THCState *state, THCTensor *self, THCTensor *result) {
	THCTensor_(fftnBatchedbase)(state, self, result, CUFFT_FORWARD);
	THCTensor_(mul)(state, result, result, ccx(1 / sqrt(THCTensor_(nElement)(state, result)),0));
}

void THCTensor_(ifftnBatched)(THCState *state, THCTensor *self, THCTensor *result) {
	THCTensor_(fftnBatchedbase)(state, self, result, CUFFT_INVERSE);
	THCTensor_(mul)(state, result, result, ccx(1 / sqrt(THCTensor_(nElement)(state, result)),0));
}

void THCTensor_(fft)(THCState *state, THCTensor *result, THCTensor *self) {
	THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, result, self));
	if (self != result)
		THCTensor_(resizeAs)(state, result, self);
	if(THCTensor_(nDimension)(state, self) < 1)
		THError("tensor must at least have dimension 1\n");
	int self_ndim = THCTensor_(nDimension)(state, self);
	if (!THCTensor_(isSameSizeAs)(state, self, result))
    THError("self_ndim must be equal result_ndim\n");
	int self_batch_dim = 1;
	for(int i = 0; i< self_ndim-1; i++){
		self_batch_dim *= THCTensor_(size)(state, self, i);
	}

	THLongStorage *new_self_size = THLongStorage_newWithSize2( self_batch_dim, THCTensor_(size)(state, self, self_ndim-1));
	THCTensor *new_self = THCTensor_(newView)(state, self, new_self_size);

	THLongStorage *new_result_size = THLongStorage_newWithSize2( self_batch_dim, THCTensor_(size)(state, self, self_ndim-1));
	THCTensor *new_result = THCTensor_(newView)(state, result, new_result_size);

	THCTensor_(fftnBatched)(state,new_self,new_result);
	THLongStorage_free(new_self_size);
	THLongStorage_free(new_result_size);
	THCTensor_(free)(state,new_result);
	THCTensor_(free)(state,new_self);
}
void THCTensor_(fft2)(THCState *state, THCTensor *result, THCTensor *self) {
	THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, result, self));
	if (self != result)
		THCTensor_(resizeAs)(state, result, self);
	if(THCTensor_(nDimension)(state, self) < 2)
		THError("tensor must at least have dimension 2\n");
	int self_ndim = THCTensor_(nDimension)(state, self);
	int res_ndim = THCTensor_(nDimension)(state, result);
	// fprintf(f,"(self_dim,res_dim) = (%d,%d)\n",self_ndim,res_ndim);
	if (!THCTensor_(isSameSizeAs)(state, self, result))
    THError("self_ndim must be equal result_ndim\n");
	int self_batch_dim = 1;
	for(int i = 0; i< self_ndim-2; i++){
		self_batch_dim *= THCTensor_(size)(state, self, i);
	}
	//fprintf(f,"self_batch_dim = %d\n",self_batch_dim);
  //fprintf(f,"dim1 = %d\n",THCTensor_(size)(state, self, self_ndim-2));
	//fprintf(f,"dim2 = %d\n",THCTensor_(size)(state, self, self_ndim-1));
	THLongStorage *new_self_size = THLongStorage_newWithSize3( self_batch_dim, THCTensor_(size)(state, self, self_ndim-2),THCTensor_(size)(state, self, self_ndim-1));
	// fprintf(f,"after THLongStorage_newWithSize3\n");
	THCTensor *new_self = THCTensor_(newView)(state, self, new_self_size);
	// fprintf(f,"after newView\n");

	THLongStorage *new_result_size = THLongStorage_newWithSize3( self_batch_dim, THCTensor_(size)(state, self, self_ndim-2),THCTensor_(size)(state, self, self_ndim-1));
	// fprintf(f,"after THLongStorage_newWithSize3\n");
	THCTensor *new_result = THCTensor_(newView)(state, result, new_result_size);
	// fprintf(f,"after newView\n");
  // fclose(f);
	THCTensor_(fftnBatched)(state,new_self,new_result);
	THLongStorage_free(new_self_size);
	THLongStorage_free(new_result_size);
	THCTensor_(free)(state,new_result);
	THCTensor_(free)(state,new_self);
}

void THCTensor_(fft3)(THCState *state, THCTensor *result, THCTensor *self) {
	THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, result, self));
	if (self != result)
		THCTensor_(resizeAs)(state, result, self);
	if(THCTensor_(nDimension)(state, self) < 3)
		THError("tensor must at least have dimension 3\n");
	int self_ndim = THCTensor_(nDimension)(state, self);
	if (!THCTensor_(isSameSizeAs)(state, self, result))
    THError("self_ndim must be equal result_ndim\n");
	int self_batch_dim = 1;
	for(int i = 0; i< self_ndim-3; i++){
		self_batch_dim *= THCTensor_(size)(state, self, i);
	}

	THLongStorage *new_self_size = THLongStorage_newWithSize4( self_batch_dim, THCTensor_(size)(state, self, self_ndim-3),THCTensor_(size)(state, self, self_ndim-2),THCTensor_(size)(state, self, self_ndim-1));
	THCTensor *new_self = THCTensor_(newView)(state, self, new_self_size);

	THLongStorage *new_result_size = THLongStorage_newWithSize4( self_batch_dim, THCTensor_(size)(state, self, self_ndim-3),THCTensor_(size)(state, self, self_ndim-2),THCTensor_(size)(state, self, self_ndim-1));
	THCTensor *new_result = THCTensor_(newView)(state, result, new_result_size);

	THCTensor_(fftnBatched)(state,new_self,new_result);
	THLongStorage_free(new_self_size);
	THLongStorage_free(new_result_size);
	THCTensor_(free)(state,new_result);
	THCTensor_(free)(state,new_self);
}

void THCTensor_(fftn)(THCState *state, THCTensor *result, THCTensor *self) {
	THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, result, self));
	if (self != result)
		THCTensor_(resizeAs)(state, result, self);
	THCTensor_(fftnbase)(state, self, result, CUFFT_FORWARD);
	THCTensor_(mul)(state, result, result, ccx(1 / sqrt(THCTensor_(nElement)(state, result)),0));
}

void THCTensor_(ifft)(THCState *state, THCTensor *result, THCTensor *self) {
	THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, result, self));
	if (self != result)
		THCTensor_(resizeAs)(state, result, self);
	if(THCTensor_(nDimension)(state, self) < 1)
		THError("tensor must at least have dimension 1\n");
	int self_ndim = THCTensor_(nDimension)(state, self);
	if (!THCTensor_(isSameSizeAs)(state, self, result))
    THError("self_ndim must be equal result_ndim\n");
	int self_batch_dim = 1;
	for(int i = 0; i< self_ndim-1; i++){
		self_batch_dim *= THCTensor_(size)(state, self, i);
	}

	THLongStorage *new_self_size = THLongStorage_newWithSize2( self_batch_dim, THCTensor_(size)(state, self, self_ndim-1));
	THCTensor *new_self = THCTensor_(newView)(state, self, new_self_size);

	THLongStorage *new_result_size = THLongStorage_newWithSize2( self_batch_dim, THCTensor_(size)(state, self, self_ndim-1));
	THCTensor *new_result = THCTensor_(newView)(state, result, new_result_size);

	THCTensor_(ifftnBatched)(state,new_self,new_result);
	THLongStorage_free(new_self_size);
	THLongStorage_free(new_result_size);
	THCTensor_(free)(state,new_result);
	THCTensor_(free)(state,new_self);
}

void THCTensor_(ifft2)(THCState *state, THCTensor *result, THCTensor *self) {
	THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, result, self));
	if (self != result)
		THCTensor_(resizeAs)(state, result, self);
	if(THCTensor_(nDimension)(state, self) < 2)
		THError("tensor must at least have dimension 2\n");
	int self_ndim = THCTensor_(nDimension)(state, self);
	if (!THCTensor_(isSameSizeAs)(state, self, result))
    THError("self_ndim must be equal result_ndim\n");
	int self_batch_dim = 1;
	for(int i = 0; i< self_ndim-2; i++){
		self_batch_dim *= THCTensor_(size)(state, self, i);
	}

	THLongStorage *new_self_size = THLongStorage_newWithSize3( self_batch_dim, THCTensor_(size)(state, self, self_ndim-2),THCTensor_(size)(state, self, self_ndim-1));
	THCTensor *new_self = THCTensor_(newView)(state, self, new_self_size);

	THLongStorage *new_result_size = THLongStorage_newWithSize3( self_batch_dim, THCTensor_(size)(state, self, self_ndim-2),THCTensor_(size)(state, self, self_ndim-1));
	THCTensor *new_result = THCTensor_(newView)(state, result, new_result_size);

	THCTensor_(ifftnBatched)(state,new_self,new_result);
	THLongStorage_free(new_self_size);
	THLongStorage_free(new_result_size);
	THCTensor_(free)(state,new_result);
	THCTensor_(free)(state,new_self);
}

void THCTensor_(ifft3)(THCState *state, THCTensor *result, THCTensor *self) {
	THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, result, self));
	if (self != result)
		THCTensor_(resizeAs)(state, result, self);
	if(THCTensor_(nDimension)(state, self) < 3)
		THError("tensor must at least have dimension 3\n");
	int self_ndim = THCTensor_(nDimension)(state, self);
	if (!THCTensor_(isSameSizeAs)(state, self, result))
    THError("self_ndim must be equal result_ndim\n");
	int self_batch_dim = 1;
	for(int i = 0; i< self_ndim-3; i++){
		self_batch_dim *= THCTensor_(size)(state, self, i);
	}

	THLongStorage *new_self_size = THLongStorage_newWithSize4( self_batch_dim, THCTensor_(size)(state, self, self_ndim-3),THCTensor_(size)(state, self, self_ndim-2),THCTensor_(size)(state, self, self_ndim-1));
	THCTensor *new_self = THCTensor_(newView)(state, self, new_self_size);

	THLongStorage *new_result_size = THLongStorage_newWithSize4( self_batch_dim, THCTensor_(size)(state, self, self_ndim-3),THCTensor_(size)(state, self, self_ndim-2),THCTensor_(size)(state, self, self_ndim-1));
	THCTensor *new_result = THCTensor_(newView)(state, result, new_result_size);

	THCTensor_(ifftnBatched)(state,new_self,new_result);
	THLongStorage_free(new_self_size);
	THLongStorage_free(new_result_size);
	THCTensor_(free)(state,new_result);
	THCTensor_(free)(state,new_self);
}

void THCTensor_(ifftn)(THCState *state, THCTensor *result, THCTensor *self) {
	THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, result, self));
	if (self != result)
		THCTensor_(resizeAs)(state, result, self);
	THCTensor_(fftnbase)(state, self, result, CUFFT_INVERSE);
	THCTensor_(mul)(state, result, result, ccx(1 / sqrt(THCTensor_(nElement)(state, result)),0));
}

#endif
#endif
