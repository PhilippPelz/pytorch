#include "THCBlas.h"
#include "THCGeneral.h"
#include "THCHalf.h"

static inline ccx toCcx(hipComplex val) {
	return ccx(hipCrealf(val), hipCimagf(val));
}
static inline ccx toCcx(hipDoubleComplex val) {
	return zcx(hipCreal(val), hipCimag(val));
}

float THCudaBlas_Sdot(THCState *state, long n, float *x, long incx, float *y, long incy)
{
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  if ((n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX)) {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    float result;
    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasSdot(handle, i_n, x, i_incx, y, i_incy, &result));
    return result;
  }

  THError("Cublas_Sdot only supports n, incx and incy "
          "up to signed integer limits: %d", INT_MAX);
  return 0;
}

double THCudaBlas_Ddot(THCState *state, long n, double *x, long incx, double *y, long incy)
{
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  if ((n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX)) {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    double result;
    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasDdot(handle, i_n, x, i_incx, y, i_incy, &result));
    return result;
  }

  THError("Cublas_Ddot only supports n, incx and incy "
          "up to signed integer limits: %d", INT_MAX);
  return 0;
}

ccx THCudaBlas_Cdot(THCState *state, long n, ccx *x, long incx, ccx *y, long incy)
{
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  if ((n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX)) {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    hipFloatComplex result;
    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasCdotc(handle, i_n, reinterpret_cast<hipFloatComplex*>(x), i_incx, reinterpret_cast<hipFloatComplex*>(y), i_incy, &result));
    return toCcx(result);
  }

  THError("cublasCdot only supports n, incx and incy "
          "up to signed integer limits: %d", INT_MAX);
  return ccx(0,0);
}

zcx THCudaBlas_Zdot(THCState *state, long n, zcx *x, long incx, zcx *y, long incy)
{
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  if ((n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX)) {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    hipDoubleComplex result;
    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    hipDoubleComplex* xc = reinterpret_cast<hipDoubleComplex*>(x);
    hipDoubleComplex* yc = reinterpret_cast<hipDoubleComplex*>(y);
    THCublasCheck(hipblasZdotc(handle, i_n, xc, i_incx, yc, i_incy, &result));
    return toCcx(result);
  }

  THError("cublasZdot only supports n, incx and incy "
          "up to signed integer limits: %d", INT_MAX);
  return zcx(0,0);
}

#ifdef CUDA_HALF_TENSOR
float THCudaBlas_Hdot(THCState *state, long n, half *x, long incx, half *y, long incy)
{
#if CUDA_VERSION >= 8000
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  if ((n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX)) {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    float result;
    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasDotEx(handle, i_n, x, HIP_R_16F, i_incx, y, HIP_R_16F, i_incy, &result, HIP_R_32F, HIP_R_32F));
    return result;
}

  THError("Cublas_Hdot only supports n, incx and incy "
          "up to signed integer limits: %d", INT_MAX);
  return 0;
#else
  THError("Cublas_Hdot requires CUDA 8.0+");
  return 0;
#endif
}
#endif

/* Level 2 */
void THCudaBlas_Sgemv(THCState *state, char trans, long m, long n, float alpha, float *a, long lda, float *x, long incx, float beta, float *y, long incy)
{
  if(n == 1)
    lda = m;

  hipblasOperation_t op;
  if (trans == 't') op = HIPBLAS_OP_T;
  else if (trans == 'n') op = HIPBLAS_OP_N;
  else if (trans == 'c') op = HIPBLAS_OP_C;

  if( (m <= INT_MAX) && (n <= INT_MAX) &&
      (lda > 0) && (lda <= INT_MAX) &&
      (incx > 0) && (incx <= INT_MAX) &&
      (incy > 0) && (incy <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_lda = (int)lda;
    int i_incx = (int)incx;
    int i_incy = (int)incy;

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasSgemv(handle, op, i_m, i_n, &alpha, a, i_lda, x, i_incx, &beta, y, i_incy));
    return;
  }
  THError("Cublas_Sgemv only supports m, n, lda, incx, incy"
          "in the range 0 < [val] <= %d", INT_MAX);
}

void THCudaBlas_Dgemv(THCState *state, char trans, long m, long n, double alpha, double *a, long lda, double *x, long incx, double beta, double *y, long incy)
{
  if(n == 1)
    lda = m;

  hipblasOperation_t op;
  if (trans == 't') op = HIPBLAS_OP_T;
  else if (trans == 'n') op = HIPBLAS_OP_N;
  else if (trans == 'c') op = HIPBLAS_OP_C;

  if( (m <= INT_MAX) && (n <= INT_MAX) &&
      (lda > 0) && (lda <= INT_MAX) &&
      (incx > 0) && (incx <= INT_MAX) &&
      (incy > 0) && (incy <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_lda = (int)lda;
    int i_incx = (int)incx;
    int i_incy = (int)incy;

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasDgemv(handle, op, i_m, i_n, &alpha, a, i_lda, x, i_incx, &beta, y, i_incy));
    return;
  }
  THError("Cublas_Dgemv only supports m, n, lda, incx, incy"
          "in the range 0 < [val] <= %d", INT_MAX);
}

void THCudaBlas_Cgemv(THCState *state, char trans, long m, long n,
                              ccx alpha, ccx *a, long lda,
                              ccx *x, long incx, ccx beta,
                              ccx *y, long incy)
{
  if(n == 1)
    lda = m;

  hipblasOperation_t op;
  if (trans == 't') op = HIPBLAS_OP_T;
  else if (trans == 'n') op = HIPBLAS_OP_N;
  else if (trans == 'c') op = HIPBLAS_OP_C;

  if( (m <= INT_MAX) && (n <= INT_MAX) &&
      (lda > 0) && (lda <= INT_MAX) &&
      (incx > 0) && (incx <= INT_MAX) &&
      (incy > 0) && (incy <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_lda = (int)lda;
    int i_incx = (int)incx;
    int i_incy = (int)incy;

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    hipFloatComplex* ac = reinterpret_cast<hipFloatComplex*>(a);
    hipFloatComplex* xc = reinterpret_cast<hipFloatComplex*>(x);
    hipFloatComplex* yc = reinterpret_cast<hipFloatComplex*>(y);
    hipFloatComplex* alphac = reinterpret_cast<hipFloatComplex*>(&alpha);
    hipFloatComplex* betac = reinterpret_cast<hipFloatComplex*>(&beta);
    THCublasCheck(hipblasCgemv(handle, op, i_m, i_n, alphac, ac, i_lda, xc, i_incx, betac, yc, i_incy));
    return;
  }
  THError("Cublas_Cgemv only supports m, n, lda, incx, incy"
          "in the range 0 < [val] <= %d", INT_MAX);
}

void THCudaBlas_Zgemv(THCState *state, char trans, long m, long n,
                              zcx alpha, zcx *a,
                              long lda, zcx *x, long incx,
                              zcx beta, zcx *y,
                              long incy)
{
  if(n == 1)
    lda = m;

  hipblasOperation_t op;
  if (trans == 't') op = HIPBLAS_OP_T;
  else if (trans == 'n') op = HIPBLAS_OP_N;
  else if (trans == 'c') op = HIPBLAS_OP_C;

  if( (m <= INT_MAX) && (n <= INT_MAX) &&
      (lda > 0) && (lda <= INT_MAX) &&
      (incx > 0) && (incx <= INT_MAX) &&
      (incy > 0) && (incy <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_lda = (int)lda;
    int i_incx = (int)incx;
    int i_incy = (int)incy;

    hipDoubleComplex* ac = reinterpret_cast<hipDoubleComplex*>(a);
    hipDoubleComplex* xc = reinterpret_cast<hipDoubleComplex*>(x);
    hipDoubleComplex* yc = reinterpret_cast<hipDoubleComplex*>(y);
    hipDoubleComplex* alphac = reinterpret_cast<hipDoubleComplex*>(&alpha);
    hipDoubleComplex* betac = reinterpret_cast<hipDoubleComplex*>(&beta);

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasZgemv(handle, op, i_m, i_n, alphac, ac, i_lda, xc, i_incx, betac, yc, i_incy));
    return;
  }
  THError("Cublas_Zgemv only supports m, n, lda, incx, incy"
          "in the range 0 < [val] <= %d", INT_MAX);
}

void THCudaBlas_Sger(THCState *state, long m, long n, float alpha, float *x, long incx, float *y, long incy, float *a, long lda)
{
  if(n == 1)
    lda = m;

  if( (m <= INT_MAX) && (n <= INT_MAX) && (lda <= INT_MAX)  && (incx <= INT_MAX) && (incy <= INT_MAX) )
    {
      int i_m = (int)m;
      int i_n = (int)n;
      int i_lda = (int)lda;
      int i_incx = (int)incx;
      int i_incy = (int)incy;

      hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
      hipblasSetStream(handle, THCState_getCurrentStream(state));
      THCublasCheck(hipblasSger(handle, i_m, i_n, &alpha, x, i_incx, y, i_incy, a, i_lda));
      return;
    }
  THError("Cublas_Sger only supports m, n, lda, incx, incy"
          "with the bound [val] <= %d", INT_MAX);
}

void THCudaBlas_Dger(THCState *state, long m, long n, double alpha, double *x, long incx, double *y, long incy, double *a, long lda)
{
  if(n == 1)
    lda = m;

  if( (m <= INT_MAX) && (n <= INT_MAX) && (lda <= INT_MAX)  && (incx <= INT_MAX) && (incy <= INT_MAX) )
    {
      int i_m = (int)m;
      int i_n = (int)n;
      int i_lda = (int)lda;
      int i_incx = (int)incx;
      int i_incy = (int)incy;

      hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
      hipblasSetStream(handle, THCState_getCurrentStream(state));
      THCublasCheck(hipblasDger(handle, i_m, i_n, &alpha, x, i_incx, y, i_incy, a, i_lda));
      return;
    }
  THError("Cublas_Dger only supports m, n, lda, incx, incy"
          "with the bound [val] <= %d", INT_MAX);
}
void THCudaBlas_Cger(THCState *state, long m, long n, ccx alpha,
                             ccx *x, long incx, ccx *y, long incy,
                             ccx *a, long lda)
{
  if(n == 1)
    lda = m;

  if( (m <= INT_MAX) && (n <= INT_MAX) && (lda <= INT_MAX)  && (incx <= INT_MAX) && (incy <= INT_MAX) )
    {
      int i_m = (int)m;
      int i_n = (int)n;
      int i_lda = (int)lda;
      int i_incx = (int)incx;
      int i_incy = (int)incy;

      hipFloatComplex* ac = reinterpret_cast<hipFloatComplex*>(a);
      hipFloatComplex* xc = reinterpret_cast<hipFloatComplex*>(x);
      hipFloatComplex* yc = reinterpret_cast<hipFloatComplex*>(y);
      hipFloatComplex* alphac = reinterpret_cast<hipFloatComplex*>(&alpha);

      hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
      hipblasSetStream(handle, THCState_getCurrentStream(state));
      THCublasCheck(hipblasCgerc(handle, i_m, i_n, alphac, xc, i_incx, yc, i_incy, ac, i_lda));
      return;
    }
  THError("Cublas_Sger only supports m, n, lda, incx, incy"
          "with the bound [val] <= %d", INT_MAX);
}

void THCudaBlas_Zger(THCState *state, long m, long n,
                             zcx alpha, zcx *x,
                             long incx, zcx *y, long incy,
                             zcx *a, long lda)
{
  if(n == 1)
    lda = m;

  if( (m <= INT_MAX) && (n <= INT_MAX) && (lda <= INT_MAX)  && (incx <= INT_MAX) && (incy <= INT_MAX) )
    {
      int i_m = (int)m;
      int i_n = (int)n;
      int i_lda = (int)lda;
      int i_incx = (int)incx;
      int i_incy = (int)incy;

      hipDoubleComplex* ac = reinterpret_cast<hipDoubleComplex*>(a);
      hipDoubleComplex* xc = reinterpret_cast<hipDoubleComplex*>(x);
      hipDoubleComplex* yc = reinterpret_cast<hipDoubleComplex*>(y);
      hipDoubleComplex* alphac = reinterpret_cast<hipDoubleComplex*>(&alpha);

      hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
      hipblasSetStream(handle, THCState_getCurrentStream(state));
      THCublasCheck(hipblasZgerc(handle, i_m, i_n, alphac, xc, i_incx, yc, i_incy, ac, i_lda));
      return;
    }
  THError("Cublas_Dger only supports m, n, lda, incx, incy"
          "with the bound [val] <= %d", INT_MAX);
}

hipblasOperation_t convertTransToCublasOperation(char trans) {
  if (trans == 't') return HIPBLAS_OP_T;
  else if (trans == 'n') return HIPBLAS_OP_N;
  else if (trans == 'c') return HIPBLAS_OP_C;
  else {
    THError("trans must be one of: t, n, c");
    return HIPBLAS_OP_T;
  }
}

void adjustLd(char transa, char transb, long m, long n, long k, long *lda, long *ldb, long *ldc)
{
  int transa_ = ((transa == 't') || (transa == 'T'));
  int transb_ = ((transb == 't') || (transb == 'T'));

  if(n == 1)
    *ldc = m;

  if(transa_)
  {
    if(m == 1)
      *lda = k;
  }
  else
  {
    if(k == 1)
      *lda = m;
  }

  if(transb_)
  {
    if(k == 1)
      *ldb = n;
  }
  else
  {
    if(n == 1)
      *ldb = k;
  }
}

/* Level 3 */
void THCudaBlas_Sgemm(THCState *state, char transa, char transb, long m, long n, long k, float alpha, float *a, long lda, float *b, long ldb, float beta, float *c, long ldc)
{
  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (lda <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_k = (int)k;
    int i_lda = (int)lda;
    int i_ldb = (int)ldb;
    int i_ldc = (int)ldc;

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasSgemm(handle, opa, opb, i_m, i_n, i_k, &alpha, a, i_lda, b, i_ldb, &beta, c, i_ldc));
    return;
  }
  THError("Cublas_Sgemm only supports m, n, k, lda, ldb, ldc"
          "with the bound [val] <= %d", INT_MAX);
}
void THCudaBlas_Cgemm(THCState *state, char transa, char transb, long m,
                              long n, long k, ccx alpha, ccx *a,
                              long lda, ccx *b, long ldb, ccx beta,
                              ccx *c, long ldc)
{
  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (lda <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_k = (int)k;
    int i_lda = (int)lda;
    int i_ldb = (int)ldb;
    int i_ldc = (int)ldc;

    hipFloatComplex* ac = reinterpret_cast<hipFloatComplex*>(a);
    hipFloatComplex* bc = reinterpret_cast<hipFloatComplex*>(b);
    hipFloatComplex* cc = reinterpret_cast<hipFloatComplex*>(c);
    hipFloatComplex* alphac = reinterpret_cast<hipFloatComplex*>(&alpha);
    hipFloatComplex* betac = reinterpret_cast<hipFloatComplex*>(&beta);

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasCgemm(handle, opa, opb, i_m, i_n, i_k, alphac, ac, i_lda, bc, i_ldb, betac, cc, i_ldc));
    return;
  }
  THError("Cublas_Cgemm only supports m, n, k, lda, ldb, ldc"
          "with the bound [val] <= %d", INT_MAX);
}

#ifdef CUDA_HALF_TENSOR
// In CUDA 8.0, definition of data types for sgemmex changed
#if CUDA_VERSION < 8000
#  define HIP_R_16F CUBLAS_DATA_HALF
#endif

void THCudaBlas_Hgemm(THCState *state, char transa, char transb, long m, long n, long k, half alpha, half *a, long lda, half *b, long ldb, half beta, half *c, long ldc)
{
  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (lda <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX) )
    {
      int i_m = (int)m;
      int i_n = (int)n;
      int i_k = (int)k;
      int i_lda = (int)lda;
      int i_ldb = (int)ldb;
      int i_ldc = (int)ldc;

      hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
      hipblasSetStream(handle, THCState_getCurrentStream(state));

      // Simulated Hgemm
      float fAlpha = THC_half2float(alpha);
      float fBeta = THC_half2float(beta);

#if CUDA_VERSION < 9000
      THCublasCheck(cublasSgemmEx(handle, opa, opb,
                                  i_m, i_n, i_k, &fAlpha,
                                  a, HIP_R_16F, i_lda, b, HIP_R_16F,
                                  i_ldb, &fBeta, c, HIP_R_16F, i_ldc));
#else
      hipDeviceProp_t* prop = THCState_getCurrentDeviceProperties(state);
      if (prop->major >= 5){
        THCublasCheck(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));
        THCublasCheck(hipblasGemmEx(handle, opa, opb,
                                   i_m, i_n, i_k, &fAlpha,
                                   a, HIP_R_16F, i_lda, b, HIP_R_16F,
                                   i_ldb, &fBeta, c, HIP_R_16F, i_ldc,
                                   HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP));
        THCublasCheck(hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH));
      }else{
        THCublasCheck(cublasSgemmEx(handle, opa, opb,
                                    i_m, i_n, i_k, &fAlpha,
                                    a, HIP_R_16F, i_lda, b, HIP_R_16F,
                                    i_ldb, &fBeta, c, HIP_R_16F, i_ldc));
      }
#endif
      return;
    }
  THError("Cublas_Hgemm only supports m, n, k, lda, ldb, ldc"
          "with th bound [val] <= %d", INT_MAX);
}
#endif

void THCudaBlas_Dgemm(THCState *state, char transa, char transb, long m, long n, long k, double alpha, double *a, long lda, double *b, long ldb, double beta, double *c, long ldc)
{
  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (lda <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_k = (int)k;
    int i_lda = (int)lda;
    int i_ldb = (int)ldb;
    int i_ldc = (int)ldc;

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasDgemm(handle, opa, opb, i_m, i_n, i_k, &alpha, a, i_lda, b, i_ldb, &beta, c, i_ldc));
    return;
  }
  THError("Cublas_Dgemm only supports m, n, k, lda, ldb, ldc"
          "with the bound [val] <= %d", INT_MAX);
}

void THCudaBlas_Zgemm(THCState *state, char transa, char transb, long m,
                              long n, long k, zcx alpha,
                              zcx *a, long lda, zcx *b,
                              long ldb, zcx beta,
                              zcx *c, long ldc)
{
  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (lda <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_k = (int)k;
    int i_lda = (int)lda;
    int i_ldb = (int)ldb;
    int i_ldc = (int)ldc;

    hipDoubleComplex* ac = reinterpret_cast<hipDoubleComplex*>(a);
    hipDoubleComplex* bc = reinterpret_cast<hipDoubleComplex*>(b);
    hipDoubleComplex* cc = reinterpret_cast<hipDoubleComplex*>(c);
    hipDoubleComplex* alphac = reinterpret_cast<hipDoubleComplex*>(&alpha);
    hipDoubleComplex* betac = reinterpret_cast<hipDoubleComplex*>(&beta);

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasZgemm(handle, opa, opb, i_m, i_n, i_k, alphac, ac, i_lda, bc, i_ldb, betac, cc, i_ldc));
    return;
  }
  THError("Cublas_Dgemm only supports m, n, k, lda, ldb, ldc"
          "with the bound [val] <= %d", INT_MAX);
}


void THCudaBlas_SgemmBatched(THCState *state, char transa, char transb, long m, long n, long k,
                             float alpha, const float *a[], long lda, const float *b[], long ldb,
                             float beta, float *c[], long ldc, long batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_SgemmBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasSgemmBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, b, (int)ldb, &beta, c, (int)ldc,
                                   (int)batchCount));
}

#if CUDA_VERSION >= 8000
void THCudaBlas_SgemmStridedBatched(THCState *state, char transa, char transb, long m, long n, long k,
                             float alpha, const float *a, long lda, long strideA, const float *b, long ldb, long strideB,
                             float beta, float *c, long ldc, long strideC, long batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
        
  {
    THError("Cublas_SgemmStridedBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasSgemmStridedBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, strideA, b, (int)ldb, strideB, &beta, c, (int)ldc, strideC,
                                   (int)batchCount));
}
#endif

void THCudaBlas_DgemmBatched(THCState *state, char transa, char transb, long m, long n, long k,
                             double alpha, const double *a[], long lda, const double *b[], long ldb,
                             double beta, double *c[], long ldc, long batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_DgemmBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasDgemmBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, b, (int)ldb, &beta, c, (int)ldc,
                                   (int)batchCount));
}
void THCudaBlas_CgemmBatched(THCState *state, char transa, char transb,
                                     long m, long n, long k, ccx alpha,
                                     const ccx *a[], long lda,
                                     const ccx *b[], long ldb,
                                     ccx beta, ccx *c[], long ldc,
                                     long batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_CgemmBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  const hipFloatComplex** ac = reinterpret_cast<const hipFloatComplex**>(a);
  const hipFloatComplex** bc = reinterpret_cast<const hipFloatComplex**>(b);
  hipFloatComplex** cc = reinterpret_cast<hipFloatComplex**>(c);
  hipFloatComplex* alphac = reinterpret_cast<hipFloatComplex*>(&alpha);
  hipFloatComplex* betac = reinterpret_cast<hipFloatComplex*>(&beta);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasCgemmBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   alphac, ac, (int)lda, bc, (int)ldb, betac, cc, (int)ldc,
                                   (int)batchCount));
}

void THCudaBlas_ZgemmBatched(THCState *state, char transa, char transb,
                                     long m, long n, long k,
                                     zcx alpha,
                                     const zcx *a[], long lda,
                                     const zcx *b[], long ldb,
                                     zcx beta, zcx *c[],
                                     long ldc, long batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_ZgemmBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  const hipDoubleComplex** ac = reinterpret_cast<const hipDoubleComplex**>(a);
  const hipDoubleComplex** bc = reinterpret_cast<const hipDoubleComplex**>(b);
  hipDoubleComplex** cc = reinterpret_cast<hipDoubleComplex**>(c);
  hipDoubleComplex* alphac = reinterpret_cast<hipDoubleComplex*>(&alpha);
  hipDoubleComplex* betac = reinterpret_cast<hipDoubleComplex*>(&beta);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasZgemmBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   alphac, ac, (int)lda, bc, (int)ldb, betac, cc, (int)ldc,
                                   (int)batchCount));
}

#if CUDA_VERSION >= 8000
void THCudaBlas_DgemmStridedBatched(THCState *state, char transa, char transb, long m, long n, long k,
                             double alpha, const double *a, long lda, long strideA, const double *b, long ldb, long strideB,
                             double beta, double *c, long ldc, long strideC, long batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_DgemmBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasDgemmStridedBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, strideA, b, (int)ldb, strideB, &beta, c, (int)ldc, strideC, 
                                   (int)batchCount));
}
#endif

/* Inverse */
void THCudaBlas_Sgetrf(THCState *state, int n, float **a, int lda, int *pivot, int *info, int batchSize) {
  if( (n >= INT_MAX) || (lda >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Sgetrf only supports n, lda, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasSgetrfBatched(handle, n, a, lda, pivot, info, batchSize));
}

void THCudaBlas_Dgetrf(THCState *state, int n, double **a, int lda, int *pivot, int *info, int batchSize) {
  if( (n >= INT_MAX) || (lda >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Dgetrf only supports n, lda, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasDgetrfBatched(handle, n, a, lda, pivot, info, batchSize));
}

void THCudaBlas_Cgetrf(THCState *state, int n, ccx **a, int lda,
                               int *pivot, int *info, int batchSize) {
  if( (n >= INT_MAX) || (lda >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Cgetrf only supports n, lda, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  hipFloatComplex** ac = reinterpret_cast<hipFloatComplex**>(a);
  THCublasCheck(hipblasCgetrfBatched(handle, n, ac, lda, pivot, info, batchSize));
}

void THCudaBlas_Zgetrf(THCState *state, int n, zcx **a,
                               int lda, int *pivot, int *info, int batchSize) {
  if( (n >= INT_MAX) || (lda >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Zgetrf only supports n, lda, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  hipDoubleComplex** ac = reinterpret_cast<hipDoubleComplex**>(a);
  THCublasCheck(hipblasZgetrfBatched(handle, n, ac, lda, pivot, info, batchSize));
}

THC_API void THCudaBlas_Sgetrs(THCState *state, char transa, int n, int nrhs, const float **a, int lda, int *pivot, float **b, int ldb, int *info, int batchSize)
{
  if( (n >= INT_MAX) || (nrhs >= INT_MAX) || (lda >= INT_MAX) || (ldb >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Dgetrs only supports n, nrhs, lda, ldb, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }

  // no need to adjust leading dimensions, since matrices are square
  hipblasOperation_t opa = convertTransToCublasOperation(transa);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasSgetrsBatched(handle, opa, n, nrhs, a, lda, pivot, b, ldb, info, batchSize));
}


THC_API void THCudaBlas_Dgetrs(THCState *state, char transa, int n, int nrhs, const double **a, int lda, int *pivot, double **b, int ldb, int *info, int batchSize)
{
  if( (n >= INT_MAX) || (nrhs >= INT_MAX) || (lda >= INT_MAX) || (ldb >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Dgetrs only supports n, nrhs, lda, ldb, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }

  // no need to adjust leading dimensions, since matrices are square
  hipblasOperation_t opa = convertTransToCublasOperation(transa);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasDgetrsBatched(handle, opa, n, nrhs, a, lda, pivot, b, ldb, info, batchSize));
}

THC_API void THCudaBlas_Cgetrs(THCState *state, char transa, int n, int nrhs,
                               const ccx **a, int lda, int *pivot,
                               ccx **b, int ldb, int *info,
                               int batchSize)
{
  if( (n >= INT_MAX) || (nrhs >= INT_MAX) || (lda >= INT_MAX) || (ldb >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Dgetrs only supports n, nrhs, lda, ldb, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }

  // no need to adjust leading dimensions, since matrices are square
  hipblasOperation_t opa = convertTransToCublasOperation(transa);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  const hipFloatComplex** ac = reinterpret_cast<const hipFloatComplex**>(a);
  hipFloatComplex** bc = reinterpret_cast<hipFloatComplex**>(b);
  THCublasCheck(hipblasCgetrsBatched(handle, opa, n, nrhs, ac, lda, pivot, bc, ldb, info, batchSize));
}


THC_API void THCudaBlas_Zgetrs(THCState *state, char transa, int n, int nrhs,
                               const zcx **a, int lda, int *pivot,
                               zcx **b, int ldb, int *info,
                               int batchSize)
{
  if( (n >= INT_MAX) || (nrhs >= INT_MAX) || (lda >= INT_MAX) || (ldb >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Dgetrs only supports n, nrhs, lda, ldb, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }

  // no need to adjust leading dimensions, since matrices are square
  hipblasOperation_t opa = convertTransToCublasOperation(transa);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  const hipDoubleComplex** ac = reinterpret_cast<const hipDoubleComplex**>(a);
  hipDoubleComplex** bc = reinterpret_cast<hipDoubleComplex**>(b);
  THCublasCheck(hipblasZgetrsBatched(handle, opa, n, nrhs, ac, lda, pivot, bc, ldb, info, batchSize));
}

void THCudaBlas_Sgetri(THCState *state, int n, const float **a, int lda, int *pivot, float **c, int ldc, int *info, int batchSize) {

  if( (n >= INT_MAX) || (lda >= INT_MAX)|| (ldc >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Sgetri only supports n, lda, ldc, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasSgetriBatched(handle, n, a, lda, pivot, c, ldc, info, batchSize));
}

void THCudaBlas_Dgetri(THCState *state, int n, const double **a, int lda, int *pivot, double **c, int ldc, int *info, int batchSize) {

  if( (n >= INT_MAX) || (lda >= INT_MAX)|| (ldc >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Dgetri only supports n, lda, ldc, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasDgetriBatched(handle, n, a, lda, pivot, c, ldc, info, batchSize));
}

void THCudaBlas_Cgetri(THCState *state, int n, const ccx **a,
                               int lda, int *pivot, ccx **c, int ldc,
                               int *info, int batchSize) {

  if( (n >= INT_MAX) || (lda >= INT_MAX)|| (ldc >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Sgetri only supports n, lda, ldc, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  const hipFloatComplex** ac = reinterpret_cast<const hipFloatComplex**>(a);
  hipFloatComplex** cc = reinterpret_cast<hipFloatComplex**>(c);
  THCublasCheck(hipblasCgetriBatched(handle, n, ac, lda, pivot, cc, ldc, info, batchSize));
}

void THCudaBlas_Zgetri(THCState *state, int n,
                               const zcx **a, int lda, int *pivot,
                               zcx **c, int ldc, int *info,
                               int batchSize) {

  if( (n >= INT_MAX) || (lda >= INT_MAX)|| (ldc >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Dgetri only supports n, lda, ldc, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  const hipDoubleComplex** ac = reinterpret_cast<const hipDoubleComplex**>(a);
  hipDoubleComplex** cc = reinterpret_cast<hipDoubleComplex**>(c);
  THCublasCheck(hipblasZgetriBatched(handle, n, ac, lda, pivot, cc, ldc, info, batchSize));
}
