#include "hip/hip_runtime.h"
#include "THCTensorFFT.h"

inline void __cufftSafeCall(hipfftResult err, const char *file, const int line) {
	FILE *f;
	f = fopen("/home/philipp/cufftSafeCall.log", "a+");
	if (HIPFFT_SUCCESS != err) {
		fprintf(f,"CUFFT error in file '%s', line %d\n %s\nerror: %d\nterminating!\n",
		file, line, err, _cudaGetErrorEnum(err));
		hipDeviceReset();
		// assert(0);
	}
	fclose(f);
}

#include "generic/THCTensorFFT.cu"
#include "THCGenerateComplexTypes.h"
