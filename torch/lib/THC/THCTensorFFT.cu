#include "THCTensorFFT.h"
#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#ifdef _CUFFT_H_
// cuFFT API errors
static const char *_cudaGetErrorEnum(hipfftResult error)
{
	switch (error)
	{
		case HIPFFT_SUCCESS:
		return "HIPFFT_SUCCESS";

		case HIPFFT_INVALID_PLAN:
		return "HIPFFT_INVALID_PLAN";

		case HIPFFT_ALLOC_FAILED:
		return "HIPFFT_ALLOC_FAILED";

		case HIPFFT_INVALID_TYPE:
		return "HIPFFT_INVALID_TYPE";

		case HIPFFT_INVALID_VALUE:
		return "HIPFFT_INVALID_VALUE";

		case HIPFFT_INTERNAL_ERROR:
		return "HIPFFT_INTERNAL_ERROR";

		case HIPFFT_EXEC_FAILED:
		return "HIPFFT_EXEC_FAILED";

		case HIPFFT_SETUP_FAILED:
		return "HIPFFT_SETUP_FAILED";

		case HIPFFT_INVALID_SIZE:
		return "HIPFFT_INVALID_SIZE";

		case HIPFFT_UNALIGNED_DATA:
		return "HIPFFT_UNALIGNED_DATA";

		case HIPFFT_INCOMPLETE_PARAMETER_LIST:
		return "HIPFFT_INCOMPLETE_PARAMETER_LIST";

    case HIPFFT_INVALID_DEVICE:
    return "HIPFFT_INVALID_DEVICE";

    case HIPFFT_PARSE_ERROR:
    return "HIPFFT_PARSE_ERROR";

    case HIPFFT_NO_WORKSPACE:
    return "HIPFFT_NO_WORKSPACE";

    case HIPFFT_NOT_IMPLEMENTED:
    return "HIPFFT_NOT_IMPLEMENTED";

    case CUFFT_LICENSE_ERROR:
    return "CUFFT_LICENSE_ERROR";

    case HIPFFT_NOT_SUPPORTED:
    return "HIPFFT_NOT_SUPPORTED";
	}

	return "<unknown>";
}
#endif

inline void __cufftSafeCall(hipfftResult err, const char *file, const int line) {
	if (HIPFFT_SUCCESS != err) {
		fprintf(stderr,"CUFFT error in file '%s', line %d\n %d\nerror: %s\nterminating!\n",
				file, line, err, _cudaGetErrorEnum(err));
		hipDeviceReset();
	}
}

#include "generic/THCTensorFFT.cu"
#include "THCGenerateComplexTypes.h"
